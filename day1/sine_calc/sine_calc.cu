#include "hip/hip_runtime.h"
__constant__ float PI_NUMBER;

// The kernel to be executed in many threads
__global__ void sine_kernel ( float Period, float * result )
{
    // Global thread index
    // ...
	
    // Do the calculations, corresponding to the thread. Use PI_NUMBER constant!
    // ...
}

#include <stdio.h>

int sine_device( float Period, size_t n, float *result )
{

    // Allocate memory on GPU
    // cuerr = hipMalloc(...

    // Send PI number to constant memory PI_NUMBER
    // cuerr = hipMemcpyToSymbol (...
    
    // Set up the kernel launch configuration for n threads 
    // (note BLOCK_SIZE is a pre-defined macro value!)
   // dim3 threads = ...
   // dim3 blocks  = ...

    // Launch the kernel using the configuration set up before
    // ...    

    // Wait the kernel to be finished (hipDeviceSynchronize)
    // ...

    // Copy the results back to CPU memory
    // cuerr = hipMemcpy (...

    // Free GPU memory
    // hipFree (...

    return 0;
}

#include <malloc.h>
#include <stdlib.h>

float original_function(int i, float Period) {
  return sinf(2.0f * float(M_PI) / Period * float(i));
} 

int main ( int argc, char* argv[] )
{
    if (argc != 2)
    {
        printf("Usage: %s <n>\n", argv[0]);
        printf("Where n must be a multiplier of %d\n", BLOCK_SIZE);
        return 0;
    }

    int n = atoi(argv[1]), nb = n * sizeof(float);
    printf("n = %d\n", n);
    if (n <= 0)
    {
        fprintf(stderr, "Invalid n: %d, must be positive\n", n);
        return 1;
    }
    if (n % BLOCK_SIZE)
    {
        fprintf(stderr, "Invalid n: %d, must be a multiplier of %d\n",
            n, BLOCK_SIZE);
        return 1;
    }

    float Period = 256.0f;

    float * result = (float*)malloc(nb);

    int status = sine_device (Period, n, result);
    if (status) return status;

    int imaxdiff = 0;
    float maxdiff = 0.0f;
    float maxdiff_good = 0.0f;
    float maxdiff_bad = 0.0f;
    for (int i = 0; i < n; i++)
    {
        float gold = original_function(i, Period); 
        float diff = result[i] / gold;
        if (diff != diff) diff = 0; else diff = 1.0 - diff;
        if (diff > maxdiff)
        {
            maxdiff = diff;
            imaxdiff = i;
            maxdiff_good = gold;
            maxdiff_bad = result[i];
        }
    }
    printf("Max diff = %f% @ i = %d: %f != %f\n",
        maxdiff * 100, imaxdiff, maxdiff_bad, maxdiff_good);
    return 0;
}

